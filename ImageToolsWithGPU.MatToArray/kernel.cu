#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#include "kernel.h"

__global__ void addKernel(int* c, const int* a, const int* b)
{
	int i = threadIdx.x;
	c[i] = a[i] + b[i];
}

hipError_t CopyDataToDevice(unsigned frameCount, unsigned char** allImageDataOnHost, unsigned char** allImageDataOnDevice, unsigned width, unsigned height)
{
	auto cudaStatus = hipSuccess;
	for (auto i = 0; i < frameCount; ++i)
	{
		cudaStatus = hipMemcpy(allImageDataOnDevice[i], allImageDataOnHost[i], width * height, hipMemcpyHostToDevice);
		if (cudaStatus != hipSuccess)
		{
			return cudaStatus;
		}
	}
	return cudaStatus;
}